
#include <hip/hip_runtime.h>
extern "C"
__global__ void kMul(double* a, double* b, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = a[idx] * b[idx];
  }
}


extern "C"
__global__ void kFillArray(double* a, int m, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = a[idx % m];
  }
}


extern "C"
__global__ void kFill(double v, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = v;
  }
}

extern "C"
__global__ void kSigmoid(double* a, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = 1/(1+__expf(-1*a[idx]));
  }
}
