
#include <hip/hip_runtime.h>
extern "C"
__global__ void kMul(double* a, double* b, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = a[idx] * b[idx];
  }
}


extern "C"
__global__ void kFillArray(double* a, int m, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = a[idx % m];
  }
}


extern "C"
__global__ void kFill(double v, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = v;
  }
}

extern "C"
__global__ void kSigmoid(double* a, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = 1/(1+ exp(-1*a[idx]));
  }
}

extern "C"
__global__ void kPow(double* a, double y, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    a[idx] = pow(a[idx], y);
  }
}

extern "C"
__global__ void kInverseElements(double* a, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    a[idx] = (a[idx]==0.0)?0.0:1.0/a[idx];
  }
}

extern "C"
__global__ void kSqrt(double* a, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    a[idx] = sqrt(a[idx]);
  }
}

extern "C"
__global__ void kDivByColumnVector(double *a, int m, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = (a[idx/m]==0.0)?0.0:dest[idx]/a[idx/m];
  }
}

extern "C"
__global__ void kMulByColumnVector(double *a, int m, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = dest[idx]*a[idx/m];
  }
}
