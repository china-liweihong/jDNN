
#include <hip/hip_runtime.h>
extern "C"

__global__ void kMul(double* a, double* b, double* dest, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<n) {
    dest[idx] = a[idx] * b[idx];
  }
}
